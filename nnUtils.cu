#include "hip/hip_runtime.h"
#include "nnUtils.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
void printVec(int** a, int n);

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
            file, line );
    exit( EXIT_FAILURE );
    }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                              hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

int listMax(int numValues, int *values)
{
    int max = 0;
    for (int i = 0; i < numValues; i++)
    {
        if (values[i] > max)
        {
            max = values[i];
        }
    }
    return max;
}

__device__ int d_listMax(int numValues, int *values)
{
    int max = 0;
    for (int i = 0; i < numValues; i++)
    {
        if (values[i] > max)
        {
            max = values[i];
        }
    }
    return max;
}

float *createNetwork(int numLayers, int *layerSizes)
{

    int numWeights = getNumNetworkWeights(numLayers, layerSizes);
    float *weights = (float *)malloc(sizeof(float) * numWeights);
    int flatIndex = 0;
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex++)
        {
            for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1] + 1; weightIndex++)
            {
                weights[flatIndex] = (float)flatIndex;
                flatIndex ++;
            }
        }
    }
    return weights;
}

void printNetwork(float *weights, int numLayers, int *layerSizes)
{
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex++)
    {
        printf("\n---<Layer %d>\n", layerIndex);
        printf("Layer size: %d\n", layerSizes[layerIndex]);
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex++)
        {
            printf("[node %d] ", nodeIndex);
            if (layerIndex > 0)
            {
                // print weights pointing to this node
                int numWeights = layerSizes[layerIndex - 1] + 1;
                for (int weightIndex = 0; weightIndex < numWeights; weightIndex ++)
                {
                    int index = getIndex(
                        layerIndex, nodeIndex, weightIndex,
                        layerSizes
                    );
                    printf("%.3f, ", weights[index]);
                    if (weightIndex == numWeights - 1)
                    {
                        printf("(bias)");
                    }
                }
            }
            printf("\n");
        }
        printf("\n");
    }
}

void initNetworkWeights(float *weights, int numLayers, int *layerSizes)
{
    srand(time(NULL));
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
        {
            int numWeights = 1 + layerSizes[layerIndex - 1];
            for (int weightIndex = 0; weightIndex < numWeights; weightIndex ++)
            {
                int index = getIndex(
                    layerIndex, nodeIndex, weightIndex,
                    layerSizes
                );
                weights[index] = (float)((rand() % 10000 + 1 - 5000)) / 10000.0f;
            }
        }
    }
}

int __device__ __host__ getIndex(int layerIndex, int nodeIndex, int weightIndex, int *layerSizes)
{
    assert(layerIndex > 0);

    int prevWeights = 0;
    for (int l = 1; l < layerIndex; l++)
    {
        prevWeights = prevWeights + layerSizes[l] * (layerSizes[l-1] + 1);
    }
    return prevWeights + nodeIndex * (layerSizes[layerIndex - 1] + 1) + weightIndex;
}

__device__ __host__ int getNumNetworkWeights(int numLayers, int *layerSizes)
{
    int w = 0;
    for (int l = 1; l < numLayers; l ++)
    {
        w = w + layerSizes[l] * (layerSizes[l - 1] + 1);
    }
    return w;
}

__device__ __host__ int getNumValueNodes(int numLayers, int *layerSizes)
{
    int numValues = 0;
    for (int l = 0; l < numLayers; l ++)
    {
        numValues = numValues + layerSizes[l];
    }
    return numValues;
}

__device__ __host__ int getValueIndex(int *layerSizes, int layerIndex, int nodeIndex)
{
    int numPrev = 0;
    for (int l = 1; l <= layerIndex; l++)
    {
        numPrev = numPrev + layerSizes[l-1];
    }
    return numPrev + nodeIndex;
}

__device__ __host__ int getNumErrorNodes(int numLayers, int *layerSizes)
{
    int numErrors = 0;
    for (int l = 1; l < numLayers; l ++) // don't count input layer because it can't have errors
    {
        numErrors = numErrors + layerSizes[l];
    }
    return numErrors;
}

__device__ __host__ int getErrorIndex(int *layerSizes, int layerIndex, int nodeIndex)
{
    int numPrev = 0;
    for (int l = 2; l <= layerIndex; l++) // don't count input layer because it can't have errors
    {
        numPrev = numPrev + layerSizes[l-1];
    }
    return numPrev + nodeIndex;
}

__global__ void trainNetworkGpu(float *weights, int numLayers, int *layerSizes,
    float *trainingData, int numTrainingData,
    int numIterations, float *trueValues, float learnRate, float *weightDeltas,
    float *nodeErrors, float *nodeValues, float *scratchWeights)
{

    int dataIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (dataIndex >= numTrainingData)
    {
        return;
    }

    int debug = 0;
    int numWeights = getNumNetworkWeights(numLayers, layerSizes);

    int numValueNodes = getNumValueNodes(numLayers, layerSizes);
    int numErrorNodes = getNumErrorNodes(numLayers, layerSizes);

    int myWeightsIndex = (blockIdx.x * blockDim.x + threadIdx.x) * numWeights;

    int nodeDataValuesOffset = (blockIdx.x * blockDim.x + threadIdx.x) * numValueNodes;
    int nodeDataErrorsOffset = (blockIdx.x * blockDim.x + threadIdx.x) * numErrorNodes;

    int dataStartIndex = dataIndex * layerSizes[0];
    int trueValueStartIndex = dataIndex * layerSizes[numLayers - 1];

    for (int iterationIndex = 0; iterationIndex < numIterations; iterationIndex ++)
    {
        if (debug)
        {
            printf("start loading training sample\n");
        }
        // load training sample
        for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
        {
            nodeValues[nodeDataValuesOffset + nodeIndex] = trainingData[dataStartIndex + nodeIndex];
        }

        if (debug)
        {
            printf("loaded training sample\n");
        }
        // forward compute
        // start with first hidden layer
        for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                float sum = 0;
                for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                {
                    float prevLayerValue = nodeValues[nodeDataValuesOffset + getValueIndex(layerSizes, layerIndex - 1, weightIndex)];
                    int index = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                    sum += prevLayerValue * scratchWeights[myWeightsIndex + index];
                }
                // add bias
                int biasIndex = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], layerSizes);
                sum += scratchWeights[myWeightsIndex + biasIndex];
                nodeValues[nodeDataValuesOffset + getValueIndex(layerSizes, layerIndex, nodeIndex)] = d_activationFunction(sum);
            }
        }
        // find error of layers
        for (int layerIndex = numLayers - 1; layerIndex > 0; layerIndex --)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                if (layerIndex == numLayers - 1)
                {
                    // special case for output layer
                    float value = nodeValues[nodeDataValuesOffset + getValueIndex(layerSizes, layerIndex, nodeIndex)];
                    float actual = trueValues[trueValueStartIndex + nodeIndex];
                    nodeErrors[nodeDataErrorsOffset + getErrorIndex(layerSizes, layerIndex, nodeIndex)] =
                        value *
                        (1 - value) *
                        (value - actual);
                }
                else
                {
                    float sum = 0;
                    for (int nextLayerNodeIndex = 0; nextLayerNodeIndex < layerSizes[layerIndex + 1]; nextLayerNodeIndex ++)
                    {
                        int index = getIndex(layerIndex + 1, nextLayerNodeIndex, nodeIndex, layerSizes);
                        sum += scratchWeights[myWeightsIndex + index] *
                            nodeErrors[nodeDataErrorsOffset + getErrorIndex(layerSizes, layerIndex + 1, nextLayerNodeIndex)];
                    }
                    float value = nodeValues[nodeDataValuesOffset + getValueIndex(layerSizes, layerIndex, nodeIndex)];
                    nodeErrors[nodeDataErrorsOffset + getErrorIndex(layerSizes, layerIndex, nodeIndex)] = sum * value * (1 - value);
                }
            }
        }
        if (debug)
        {
            printf("finished finding errors\n");
        }
        // update weights
        for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                {
                    int index = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                    scratchWeights[myWeightsIndex + index] -=
                        learnRate *
                        nodeErrors[nodeDataErrorsOffset + getErrorIndex(layerSizes, layerIndex, nodeIndex)] *
                        nodeValues[nodeDataValuesOffset + getValueIndex(layerSizes, layerIndex - 1, weightIndex)];
                }
                // update bias
                int index = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], layerSizes);
                scratchWeights[myWeightsIndex + index] -=
                    learnRate *
                    nodeErrors[nodeDataErrorsOffset + getErrorIndex(layerSizes, layerIndex, nodeIndex)];
            }
        }
        if (debug)
        {
            printf("finished updating weights\n");
        }
    }
    if (debug)
    {
        printf("finished internal iterations\n");
    }
}

void trainNetwork(float *weights, int numLayers, int *layerSizes,
    float *trainingData, int numTrainingData,
    int numIterations, float *trueValues, float learnRate)
{
    // node delta
    float *errors = (float *)malloc(sizeof(float) * getNumErrorNodes(numLayers, layerSizes));

    // activation values
    float *values = (float *)malloc(sizeof(float) * getNumValueNodes(numLayers, layerSizes));

    for (int iterationIndex = 0; iterationIndex < numIterations; iterationIndex ++)
    {
        for (int dataIndex = 0; dataIndex < numTrainingData; dataIndex ++)
        {
            if (dataIndex % 1000 == 0)
            {
                printf("sample %d\n", dataIndex);
            }

            int dataStartIndex = dataIndex * layerSizes[0];
            int trueValueStartIndex = dataIndex * layerSizes[numLayers - 1];

            // load training sample
            for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
            {
                values[nodeIndex] = trainingData[dataStartIndex + nodeIndex];
            }

            // forward compute
            // start with first hidden layer
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    float sum = 0;
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                    {
                        float prevLayerValue = values[getValueIndex(layerSizes, layerIndex - 1, weightIndex)];
                        int index = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                        sum += prevLayerValue * weights[index];
                    }
                    // add bias
                    int biasIndex = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], layerSizes);
                    sum += weights[biasIndex];
                    values[getValueIndex(layerSizes, layerIndex, nodeIndex)] = activationFunction(sum);
                }
            }
            // find error of layers
            for (int layerIndex = numLayers - 1; layerIndex > 0; layerIndex --)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    if (layerIndex == numLayers - 1)
                    {
                        // special case for output layer
                        float value = values[getValueIndex(layerSizes, layerIndex, nodeIndex)];
                        float actual = trueValues[trueValueStartIndex + nodeIndex];
                        errors[getErrorIndex(layerSizes, layerIndex, nodeIndex)] =
                            value *
                            (1 - value) *
                            (value - actual);
                    }
                    else
                    {
                        float sum = 0;
                        for (int nextLayerNodeIndex = 0; nextLayerNodeIndex < layerSizes[layerIndex + 1]; nextLayerNodeIndex ++)
                        {
                            int index = getIndex(layerIndex + 1, nextLayerNodeIndex, nodeIndex, layerSizes);
                            sum += weights[index] *
                                errors[getErrorIndex(layerSizes, layerIndex + 1, nextLayerNodeIndex)];
                        }
                        float value = values[getValueIndex(layerSizes, layerIndex, nodeIndex)];
                        errors[getErrorIndex(layerSizes, layerIndex, nodeIndex)] = sum * value * (1 - value);
                    }
                }
            }

            // update weights
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                    {
                        int index = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                        weights[index] -=
                            learnRate *
                            errors[getErrorIndex(layerSizes, layerIndex, nodeIndex)] *
                            values[getValueIndex(layerSizes, layerIndex - 1, weightIndex)];
                    }
                    // update bias
                    int index = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], layerSizes);
                    weights[index] -=
                        learnRate *
                        errors[getErrorIndex(layerSizes, layerIndex, nodeIndex)];
                }
            }
        }
    }
}

float *classify(float *weights, int numLayers, int *layerSizes, float *samples, int sampleIndex)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    float *values = (float *)malloc(sizeof(float) * getNumValueNodes(numLayers, layerSizes));
    int sampleOffset = layerSizes[0] * sampleIndex;
    // load input layer from provided sample
    for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
    {
        values[nodeIndex] = samples[sampleOffset + nodeIndex];
    }
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
        {
            float sum = 0;
            for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
            {
                float prevLayerValue = values[getValueIndex(layerSizes, layerIndex - 1, weightIndex)];
                int index = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                sum += prevLayerValue * weights[index];
            }
            // add bias
            int index = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], layerSizes);
            sum += weights[index];
            values[getValueIndex(layerSizes, layerIndex, nodeIndex)] = activationFunction(sum);
        }
    }
    float *out = (float *)malloc(sizeof(float) * layerSizes[numLayers - 1]);
    for (int nodeIndex = 0; nodeIndex < layerSizes[numLayers - 1]; nodeIndex ++)
    {
        out[nodeIndex] = values[getValueIndex(layerSizes, numLayers - 1, nodeIndex)];
    }
    return out;
}

float activationFunction(float x)
{
    return (float)(1.0f / (1 + exp(x * (-1))));
}

__device__ float d_activationFunction(float x)
{
    return (float)(1.0f / (1 + exp(x * (-1))));
}

void batchTrainNetworkGpu(
    float *weights, int numLayers, int *layerSizes,
    float *trainData, int trainDataCount, int internalIterations,
    float *trueValues, float learnRate, int batchSize,
    int numEpochs, imageTrainingSamples *testCases)
{
    int numWeights = getNumNetworkWeights(numLayers, layerSizes);
    float *weightDeltas = (float *) malloc(sizeof(float) * numWeights);
    float *scratchWeights = (float *) malloc(sizeof(float) * batchSize * numWeights);
    int inDataWidth = layerSizes[0];

    int threadsPerBlock = 8;

    float *d_weights = 0;
    int *d_layerSizes = 0;
    float *d_trainData = 0;
    float *d_trueValues = 0;
    float *d_weightDeltas = 0;
    float *d_nodeErrors = 0;
    float *d_nodeValues = 0;
    float *d_scratchWeights = 0;
    int numBatches = (int)ceil((float)trainDataCount / (float)batchSize);
    int numBlocks = (int)ceil((float)batchSize / (float)threadsPerBlock); // need to check this math

    int debug = 0;
    printf("Num Batches: %d\n", numBatches);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("Num Blocks: %d\n", numBlocks);

    hipMalloc(&d_weights, sizeof(float) * numWeights);
    hipMalloc(&d_layerSizes, sizeof(int) * numLayers);
    hipMalloc(&d_trainData, sizeof(float) * batchSize * inDataWidth);
    hipMalloc(&d_trueValues, sizeof(float) * batchSize * layerSizes[numLayers - 1]);
    hipMalloc(&d_weightDeltas, sizeof(float) * numWeights);
    hipMalloc(&d_nodeErrors, sizeof(float) * getNumErrorNodes(numLayers, layerSizes) * numBlocks * threadsPerBlock);
    hipMalloc(&d_nodeValues, sizeof(float) * getNumValueNodes(numLayers, layerSizes) * numBlocks * threadsPerBlock);
    hipMalloc(&d_scratchWeights, sizeof(float) * batchSize * numWeights);

    hipMemcpy(d_layerSizes, layerSizes, sizeof(int) * numLayers, hipMemcpyHostToDevice);

    for (int epochIndex = 0; epochIndex < numEpochs; epochIndex++)
    {

        for (int batchNumber = 0; batchNumber < numBatches; batchNumber ++)
        {
            hipMemcpy(d_weights, weights, sizeof(float) * numWeights, hipMemcpyHostToDevice);

            int trainDataStartIndex = batchNumber * batchSize * inDataWidth;
            int trueValuesStartIndex = batchNumber * batchSize * layerSizes[numLayers - 1];
            int thisBatchNumSamples = batchSize;
            if ((batchNumber + 1) * batchSize > trainDataCount)
            {
                // in this case our final batch has more capacity than the number of remaining samples
                // need to copy less data in
                thisBatchNumSamples = batchSize - ((batchNumber + 1) * batchSize - trainDataCount);
            }
            int trainDataBytesToCopy = sizeof(float) * thisBatchNumSamples * inDataWidth;
            int trueValuesBytesToCopy = sizeof(float) * thisBatchNumSamples * layerSizes[numLayers - 1];
            // copy in the samples of this batch
            hipMemcpy(d_trainData, trainData + trainDataStartIndex, trainDataBytesToCopy, hipMemcpyHostToDevice);
            hipMemcpy(d_trueValues, trueValues + trueValuesStartIndex, trueValuesBytesToCopy, hipMemcpyHostToDevice);

            if (debug)
            {
                printf("start copying scratch weights\n");
            }
            for (int sampleIndex = 0; sampleIndex < thisBatchNumSamples; sampleIndex ++)
            {
                hipMemcpy(d_scratchWeights + sampleIndex * numWeights, d_weights, numWeights * sizeof(float), hipMemcpyDeviceToDevice);
            }
            if (debug)
            {
                printf("done copying scratch weights\n");
            }
            trainNetworkGpu<<<numBlocks, threadsPerBlock>>>(
                d_weights, numLayers, d_layerSizes,
                d_trainData, thisBatchNumSamples, internalIterations,
                d_trueValues, learnRate, d_weightDeltas,
                d_nodeErrors, d_nodeValues, d_scratchWeights
            );
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );

            hipMemcpy(scratchWeights, d_scratchWeights, thisBatchNumSamples * numWeights * sizeof(float), hipMemcpyDeviceToHost);

            if (debug)
            {
                printf("start adding deltas\n");
            }
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1] + 1; weightIndex ++)
                    {
                        float delta = 0;
                        int weightFlatIndex = getIndex(layerIndex, nodeIndex, weightIndex, layerSizes);
                        for (int sampleIndex = 0; sampleIndex < thisBatchNumSamples; sampleIndex ++)
                        {
                            float thisDelta = scratchWeights[sampleIndex * numWeights + weightFlatIndex] - weights[weightFlatIndex];
                            delta += thisDelta;
                        }
                        weights[weightFlatIndex] += delta;
                    }
                }
            }
            if (debug)
            {
                printf("done adding deltas\n");
            }
            for (int i = 0; i < numWeights; i++)
            {
                weightDeltas[i] = 0;
            }
            if (1 || debug)
            {
                printf("Finished epoch %d / %d, batch %d / %d\n",
                    epochIndex, numEpochs,
                    batchNumber, numBatches);
            }
        }
        printf("finished epoch %d\n", epochIndex);
        if (testCases)
        {
            testNetwork(weights, numLayers, layerSizes, testCases);
        }
    }
}

imageTrainingSamples *getImageData(char *filename, int numItems, int startingIndex)
{
    int numInputNodes = 28 * 28;
    int numOutputNodes = 10;

    imageTrainingSamples *samples = (imageTrainingSamples *)malloc(sizeof(imageTrainingSamples));
    samples->inputSamples = (float *)malloc(sizeof(float) * numItems * numInputNodes);
    samples->trueOutput = (float *)malloc(sizeof(float) * numItems * numOutputNodes);

    FILE *fileStream = fopen(filename, "r");

    char *token;
    char line[8192];
    char sep[] = ",";
    int isHeader = 1;
    int itemIndex = 0;
    int numItemsTaken = 0;
    while (numItemsTaken < numItems && fgets(line, 8192, fileStream))
    {
        if (isHeader)
        {
            // skip CSV header
            isHeader = false;
            continue;
        }

        if (itemIndex < startingIndex)
        {
            itemIndex ++;
            continue;
        }

        // first get the label / true value
        token = strtok(line, sep);
        int trueValue = strtol(token, NULL, 10);
        for (int i = 0; i < numOutputNodes; i ++)
        {
            int index = numItemsTaken * numOutputNodes + i;
            if (i == trueValue)
            {
                samples->trueOutput[index] = 1.0f;
            }
            else
            {
                samples->trueOutput[index] = 0.0f;
            }
        }

        // now get the inputs - pixels with value from 0-255
        int tokenIndex = 0;
        while (token != NULL)
        {
            float rawValue = (float)strtol(token, NULL, 10);
            samples->inputSamples[numItemsTaken * numInputNodes + tokenIndex] = rawValue / (float)255;
            token = strtok(NULL, sep);
            tokenIndex ++;
        }
        numItemsTaken ++;
        itemIndex ++;
    }

    samples->numItems = itemIndex;

    return samples;
}

__device__ __host__ void printSampleSketch(float *pixelValues, int sampleIndex)
{
    int width = 28;
    int height = 28;

    char ch = 0;
    float value = 0;

    int sampleOffset = sampleIndex * width * height;

    for (int y = 0; y < height; y ++)
    {
        for (int x = 0; x < width; x ++)
        {
            value = pixelValues[sampleOffset + y * width + x];
            if (value > .75) {
                ch = '#';
            }
            else if (value > .5)
            {
                ch = '=';
            }
            else if (value > .25)
            {
                ch = '-';
            }
            else
            {
                ch = ' ';
            }
            printf("%c", ch);
        }
        printf("\n");
    }
}

int imageSampleTrueValue(float * trueValues, int sampleIndex)
{
    int nodesPerSample = 10;
    int startIndex = nodesPerSample * sampleIndex;

    for (int nodeIndex = 0; nodeIndex < nodesPerSample; nodeIndex ++)
    {
        if (trueValues[startIndex + nodeIndex] == 1)
        {
            return nodeIndex;
        }
    }
    return 0;
}

int imageSampleTestResult(float *trueValues, int sampleIndex, float *result)
{
    int trueValue = imageSampleTrueValue(trueValues, sampleIndex);

    int selectedValue = imageSampleResultToInt(result);

    if (trueValue == selectedValue) {
        return 1;
    }

    return 0;
}

int imageSampleResultToInt(float *result)
{
    int nodesPerSample = 10;
    int selectedValue = 0;
    float highestActivation = 0;
    for (int nodeIndex = 0; nodeIndex < nodesPerSample; nodeIndex ++)
    {
        if (result[nodeIndex] > highestActivation)
        {
            highestActivation = result[nodeIndex];
            selectedValue = nodeIndex;
        }
    }

    return selectedValue;
}

void testNetwork(float *weights, int numLayers, int *layerSizes, imageTrainingSamples *testCases)
{
    int numCorrect = 0;
    for (int testCaseIndex = 0; testCaseIndex < testCases->numItems; testCaseIndex ++)
    {
        int trueValue = imageSampleTrueValue(testCases->trueOutput, testCaseIndex);
        float *result = classify(weights, numLayers, layerSizes, testCases->inputSamples, testCaseIndex);
        int isCorrect = imageSampleTestResult(testCases->trueOutput, testCaseIndex, result);
        // printf("Actual / Result: %d / %d ", trueValue, imageSampleResultToInt(result));
        for (int i = 0; i < layerSizes[numLayers - 1]; i++)
        {
            // printf("%.3f ", result[i]);
        }
        if (isCorrect)
        {
            numCorrect ++;
            // printf("Correct");
        }
        else
        {
            // printf("NOPE");
        }
    }
    printf("Accuracy: %.2f\n", (float)numCorrect / (float) testCases->numItems);
}
