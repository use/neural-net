#include "hip/hip_runtime.h"
#include "nnUtils.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
void printVec(int** a, int n);

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
            file, line );
    exit( EXIT_FAILURE );
    }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                              hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

int listMax(int numValues, int *values)
{
    int max = 0;
    for (int i = 0; i < numValues; i++)
    {
        if (values[i] > max)
        {
            max = values[i];
        }
    }
    return max;
}

__device__ int d_listMax(int numValues, int *values)
{
    int max = 0;
    for (int i = 0; i < numValues; i++)
    {
        if (values[i] > max)
        {
            max = values[i];
        }
    }
    return max;
}

float *createNetwork(int numLayers, int *layerSizes)
{

    int maxLayerSize = listMax(numLayers, layerSizes);

    float *weights = (float *)malloc(sizeof(float) * numLayers * maxLayerSize * (maxLayerSize + 1));
    for (int layerIndex = 0; layerIndex < numLayers; layerIndex++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex++)
        {
            if (layerIndex == 0)
            {
                // don't do anything because layer 0 is inputs and has no incoming weights
            }
            else
            {
                int numWeights = 1 + layerSizes[layerIndex - 1]; // +1 for bias
                for (int weightIndex = 0; weightIndex < numWeights; weightIndex++)
                {
                    int index = getIndex(
                        layerIndex, nodeIndex, weightIndex,
                        maxLayerSize
                    );
                    weights[index] = 0.5f;
                }
            }
        }
    }
    return weights;
}

void printNetwork(float *weights, int numLayers, int *layerSizes)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex++)
    {
        printf("\n---<Layer %d>\n", layerIndex);
        printf("Layer size: %d\n", layerSizes[layerIndex]);
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex++)
        {
            printf("[node %d] ", nodeIndex);
            if (layerIndex > 0)
            {
                // print weights pointing to this node
                int numWeights = layerSizes[layerIndex - 1] + 1;
                for (int weightIndex = 0; weightIndex < numWeights; weightIndex ++)
                {
                    int index = getIndex(
                        layerIndex, nodeIndex, weightIndex,
                        maxLayerSize
                    );
                    printf("%.3f, ", weights[index]);
                    if (weightIndex == numWeights - 1)
                    {
                        printf("(bias)");
                    }
                }
            }
            printf("\n");
        }
        printf("\n");
    }
}

void initNetworkWeights(float *weights, int numLayers, int *layerSizes)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    srand(time(NULL));
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
        {
            int numWeights = 1 + layerSizes[layerIndex - 1];
            for (int weightIndex = 0; weightIndex < numWeights; weightIndex ++)
            {
                int index = getIndex(
                    layerIndex, nodeIndex, weightIndex,
                    maxLayerSize
                );
                weights[index] = (float)((rand() % 10000 + 1 - 5000)) / 10000.0f;
            }
        }
    }
}

int getIndex(int layerIndex, int nodeIndex, int weightIndex, int maxLayerSize)
{
    return layerIndex * (maxLayerSize + 1) * maxLayerSize +
        nodeIndex * (maxLayerSize + 1) +
        weightIndex;
}

__device__ int d_getIndex(int layerIndex, int nodeIndex, int weightIndex, int maxLayerSize)
{
    return layerIndex * (maxLayerSize + 1) * maxLayerSize +
        nodeIndex * (maxLayerSize + 1) +
        weightIndex;
}

__global__ void trainNetworkGpu(float *weights, int numLayers, int *layerSizes,
    float *trainingData, int numTrainingData,
    int numIterations, float *trueValues, float learnRate, float *weightDeltas,
    float *nodeErrors, float *nodeValues, float *scratchWeights)
{

    int dataIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (dataIndex >= numTrainingData)
    {
        return;
    }

    int debug = 0;
    int maxLayerSize = d_listMax(numLayers, layerSizes);
    int numWeights = numLayers * maxLayerSize * (maxLayerSize + 1);
    int myWeightsIndex = (blockIdx.x * blockDim.x + threadIdx.x) * numWeights;

    int nodeDataOffset = numLayers * maxLayerSize * (blockIdx.x * blockDim.x + threadIdx.x);

    int dataStartIndex = dataIndex * layerSizes[0];
    int trueValueStartIndex = dataIndex * layerSizes[numLayers - 1];

    for (int iterationIndex = 0; iterationIndex < numIterations; iterationIndex ++)
    {
        if (debug)
        {
            printf("start loading training sample\n");
        }
        // load training sample
        for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
        {
            nodeValues[nodeDataOffset + nodeIndex] = trainingData[dataStartIndex + nodeIndex];
        }

        if (debug)
        {
            printf("loaded training sample\n");
        }
        if (debug && iterationIndex == 0 && dataIndex == 0)
        {
            printf("Training Data\n");
            for (int i = 0; i < numTrainingData; i++)
            {
                printf("[%d] ", i);
                for (int j = 0; j < layerSizes[0]; j++)
                {
                    printf("%.4f ", trainingData[dataStartIndex + j]);
                }
                printf("(");
                for (int j = 0; j < layerSizes[numLayers - 1]; j++)
                {
                    printf("%.4f ", trueValues[trueValueStartIndex + j]);
                }
                printf(")\n");
            }
            printf("Values\n");
            for (int i = 0; i < numLayers; i++)
            {
                printf("[%d] ", i);
                for (int j = 0; j < maxLayerSize; j++)
                {
                    printf("%.4f ", nodeValues[nodeDataOffset + i * maxLayerSize + j]);
                }
                printf("\n");
            }
        }
        // forward compute
        // start with first hidden layer
        for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                float sum = 0;
                for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                {
                    float prevLayerValue = nodeValues[nodeDataOffset + (layerIndex - 1) * maxLayerSize + weightIndex];
                    int index = d_getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                    sum += prevLayerValue * scratchWeights[myWeightsIndex + index];
                }
                // add bias
                int biasIndex = d_getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], maxLayerSize);
                sum += scratchWeights[myWeightsIndex + biasIndex];
                nodeValues[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex] = d_activationFunction(sum);
            }
        }
        // find error of layers
        for (int layerIndex = numLayers - 1; layerIndex > 0; layerIndex --)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                if (layerIndex == numLayers - 1)
                {
                    // special case for output layer
                    float value = nodeValues[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex];
                    float actual = trueValues[trueValueStartIndex + nodeIndex];
                    nodeErrors[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex] =
                        value *
                        (1 - value) *
                        (value - actual);
                }
                else
                {
                    float sum = 0;
                    for (int nextLayerNodeIndex = 0; nextLayerNodeIndex < layerSizes[layerIndex + 1]; nextLayerNodeIndex ++)
                    {
                        int index = d_getIndex(layerIndex + 1, nextLayerNodeIndex, nodeIndex, maxLayerSize);
                        sum += scratchWeights[myWeightsIndex + index] *
                            nodeErrors[nodeDataOffset + (layerIndex + 1) * maxLayerSize + nextLayerNodeIndex];
                    }
                    float value = nodeValues[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex];
                    nodeErrors[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex] = sum * value * (1 - value);
                }
            }
        }
        if (debug)
        {
            printf("finished finding errors\n");
        }
        // update weights
        for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
        {
            for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
            {
                for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                {
                    int index = d_getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                    scratchWeights[myWeightsIndex + index] -=
                        learnRate *
                        nodeErrors[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex] *
                        nodeValues[nodeDataOffset + (layerIndex - 1) * maxLayerSize + weightIndex];
                }
                // update bias
                int index = d_getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], maxLayerSize);
                scratchWeights[myWeightsIndex + index] -=
                    learnRate *
                    nodeErrors[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex];
            }
        }
        if (debug)
        {
            printf("finished updating weights\n");
        }
        if (
            debug && (
                iterationIndex < 11 ||
                iterationIndex == 100 ||
                iterationIndex == 1000 ||
                iterationIndex == 10000 ||
                iterationIndex == numIterations - 1
            )
        )
        {
            printf("\nIteration %d\n", iterationIndex);
            printf("(Training sample)\n");
            for (int dataNodeIndex = 0; dataNodeIndex < layerSizes[0]; dataNodeIndex ++)
            {
                printf("%.6f ", trainingData[dataStartIndex + dataNodeIndex]);
            }
            printf("\n");
            printf("(Value data below)\n");
            for (int layerIndex = 0; layerIndex < numLayers; layerIndex ++)
            {
                printf("[%d] ", layerIndex);
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    printf("%.6f ", nodeValues[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex]);
                }
                printf("\n");
            }
            printf("(Error data below)\n");
            for (int layerIndex = 0; layerIndex < numLayers; layerIndex ++)
            {
                printf("[%d] ", layerIndex);
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    printf("%.6f ", nodeErrors[nodeDataOffset + layerIndex * maxLayerSize + nodeIndex]);
                }
                printf("\n");
            }
        }
    }
    if (debug)
    {
        printf("finished internal iterations\n");
    }
}

void trainNetwork(float *weights, int numLayers, int *layerSizes,
    float *trainingData, int numTrainingData,
    int numIterations, float *trueValues, float learnRate)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    // node delta
    float errors[numLayers][maxLayerSize];
    for (int i = 0; i < numLayers; i++)
    {
        for (int j = 0; j < maxLayerSize; j++)
        {
            errors[i][j] = 0;
        }
    }

    // activation values
    float values[numLayers][maxLayerSize];
    for (int i = 0; i < numLayers; i++)
    {
        for (int j = 0; j < maxLayerSize; j++)
        {
            values[i][j] = 0;
        }
    }

    for (int iterationIndex = 0; iterationIndex < numIterations; iterationIndex ++)
    {
        for (int dataIndex = 0; dataIndex < numTrainingData; dataIndex ++)
        {
            if (dataIndex % 1000 == 0)
            {
                printf("sample %d\n", dataIndex);
            }

            int dataStartIndex = dataIndex * layerSizes[0];
            int trueValueStartIndex = dataIndex * layerSizes[numLayers - 1];

            // load training sample
            for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
            {
                values[0][nodeIndex] = trainingData[dataStartIndex + nodeIndex];
            }

            // forward compute
            // start with first hidden layer
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    float sum = 0;
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                    {
                        float prevLayerValue = values[layerIndex - 1][weightIndex];
                        int index = getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                        sum += prevLayerValue * weights[index];
                    }
                    // add bias
                    int biasIndex = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], maxLayerSize);
                    sum += weights[biasIndex];
                    values[layerIndex][nodeIndex] = activationFunction(sum);
                }
            }
            // find error of layers
            for (int layerIndex = numLayers - 1; layerIndex > 0; layerIndex --)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    if (layerIndex == numLayers - 1)
                    {
                        // special case for output layer
                        float value = values[layerIndex][nodeIndex];
                        float actual = trueValues[trueValueStartIndex + nodeIndex];
                        errors[layerIndex][nodeIndex] =
                            value *
                            (1 - value) *
                            (value - actual);
                    }
                    else
                    {
                        float sum = 0;
                        for (int nextLayerNodeIndex = 0; nextLayerNodeIndex < layerSizes[layerIndex + 1]; nextLayerNodeIndex ++)
                        {
                            int index = getIndex(layerIndex + 1, nextLayerNodeIndex, nodeIndex, maxLayerSize);
                            sum += weights[index] *
                                errors[layerIndex + 1][nextLayerNodeIndex];
                        }
                        float value = values[layerIndex][nodeIndex];
                        errors[layerIndex][nodeIndex] = sum * value * (1 - value);
                    }
                }
            }

            // update weights
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
                    {
                        int index = getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                        weights[index] -=
                            learnRate *
                            errors[layerIndex][nodeIndex] *
                            values[layerIndex - 1][weightIndex];
                    }
                    // update bias
                    int index = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], maxLayerSize);
                    weights[index] -=
                        learnRate *
                        errors[layerIndex][nodeIndex];
                }
            }
        }
    }
}

float *classify(float *weights, int numLayers, int *layerSizes, float *samples, int sampleIndex)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    float values[numLayers][maxLayerSize];
    for (int i = 0; i < numLayers; i++)
    {
        for (int j = 0; j < maxLayerSize; j++)
        {
            values[i][j] = 0;
        }
    }
    int sampleOffset = layerSizes[0] * sampleIndex;
    // load input layer from provided sample
    for (int nodeIndex = 0; nodeIndex < layerSizes[0]; nodeIndex ++)
    {
        values[0][nodeIndex] = samples[sampleOffset + nodeIndex];
    }
    for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
    {
        for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
        {
            float sum = 0;
            for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1]; weightIndex ++)
            {
                float prevLayerValue = values[layerIndex - 1][weightIndex];
                int index = getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                sum += prevLayerValue * weights[index];
            }
            // add bias
            int index = getIndex(layerIndex, nodeIndex, layerSizes[layerIndex - 1], maxLayerSize);
            sum += weights[index];
            values[layerIndex][nodeIndex] = activationFunction(sum);
        }
    }
    float *out = (float *)malloc(sizeof(float) * layerSizes[numLayers - 1]);
    for (int nodeIndex = 0; nodeIndex < layerSizes[numLayers - 1]; nodeIndex ++)
    {
        out[nodeIndex] = values[numLayers - 1][nodeIndex];
    }
    return out;
}

float activationFunction(float x)
{
    return (float)(1.0f / (1 + exp(x * (-1))));
}

__device__ float d_activationFunction(float x)
{
    return (float)(1.0f / (1 + exp(x * (-1))));
}

void batchTrainNetworkGpu(
    float *weights, int numLayers, int *layerSizes,
    float *trainData, int trainDataCount, int internalIterations,
    float *trueValues, float learnRate, int batchSize,
    int numEpochs, imageTrainingSamples *testCases)
{
    int maxLayerSize = listMax(numLayers, layerSizes);
    int numWeights = numLayers * maxLayerSize * (maxLayerSize + 1);
    float *weightDeltas = (float *) malloc(sizeof(float) * numWeights);
    float *scratchWeights = (float *) malloc(sizeof(float) * batchSize * numWeights);
    int inDataWidth = layerSizes[0];

    int threadsPerBlock = 8;

    float *d_weights = 0;
    int *d_layerSizes = 0;
    float *d_trainData = 0;
    float *d_trueValues = 0;
    float *d_weightDeltas = 0;
    float *d_nodeErrors = 0;
    float *d_nodeValues = 0;
    float *d_scratchWeights = 0;
    int numBatches = (int)ceil((float)trainDataCount / (float)batchSize);
    int numBlocks = (int)ceil((float)batchSize / (float)threadsPerBlock); // need to check this math

    int debug = 0;
    printf("Num Batches: %d\n", numBatches);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("Num Blocks: %d\n", numBlocks);

    hipMalloc(&d_weights, sizeof(float) * numWeights);
    hipMalloc(&d_layerSizes, sizeof(int) * numLayers);
    hipMalloc(&d_trainData, sizeof(float) * batchSize * inDataWidth);
    hipMalloc(&d_trueValues, sizeof(float) * batchSize * layerSizes[numLayers - 1]);
    hipMalloc(&d_weightDeltas, sizeof(float) * numWeights);
    hipMalloc(&d_nodeErrors, sizeof(float) * numLayers * maxLayerSize * numBlocks * threadsPerBlock);
    hipMalloc(&d_nodeValues, sizeof(float) * numLayers * maxLayerSize * numBlocks * threadsPerBlock);
    hipMalloc(&d_scratchWeights, sizeof(float) * batchSize * numWeights);

    hipMemcpy(d_layerSizes, layerSizes, sizeof(int) * numLayers, hipMemcpyHostToDevice);

    for (int epochIndex = 0; epochIndex < numEpochs; epochIndex++)
    {

        for (int batchNumber = 0; batchNumber < numBatches; batchNumber ++)
        {
            hipMemcpy(d_weights, weights, sizeof(float) * numWeights, hipMemcpyHostToDevice);

            int trainDataStartIndex = batchNumber * batchSize * inDataWidth;
            int trueValuesStartIndex = batchNumber * batchSize * layerSizes[numLayers - 1];
            int thisBatchNumSamples = batchSize;
            if ((batchNumber + 1) * batchSize > trainDataCount)
            {
                // in this case our final batch has more capacity than the number of remaining samples
                // need to copy less data in
                thisBatchNumSamples = batchSize - ((batchNumber + 1) * batchSize - trainDataCount);
            }
            int trainDataBytesToCopy = sizeof(float) * thisBatchNumSamples * inDataWidth;
            int trueValuesBytesToCopy = sizeof(float) * thisBatchNumSamples * layerSizes[numLayers - 1];
            // copy in the samples of this batch
            hipMemcpy(d_trainData, trainData + trainDataStartIndex, trainDataBytesToCopy, hipMemcpyHostToDevice);
            hipMemcpy(d_trueValues, trueValues + trueValuesStartIndex, trueValuesBytesToCopy, hipMemcpyHostToDevice);

            if (debug)
            {
                printf("start copying scratch weights\n");
            }
            for (int sampleIndex = 0; sampleIndex < thisBatchNumSamples; sampleIndex ++)
            {
                hipMemcpy(d_scratchWeights + sampleIndex * numWeights, d_weights, numWeights * sizeof(float), hipMemcpyDeviceToDevice);
            }
            if (debug)
            {
                printf("done copying scratch weights\n");
            }
            trainNetworkGpu<<<numBlocks, threadsPerBlock>>>(
                d_weights, numLayers, d_layerSizes,
                d_trainData, thisBatchNumSamples, internalIterations,
                d_trueValues, learnRate, d_weightDeltas,
                d_nodeErrors, d_nodeValues, d_scratchWeights
            );
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );

            hipMemcpy(scratchWeights, d_scratchWeights, thisBatchNumSamples * numWeights * sizeof(float), hipMemcpyDeviceToHost);

            if (debug)
            {
                printf("start adding deltas\n");
            }
            for (int layerIndex = 1; layerIndex < numLayers; layerIndex ++)
            {
                for (int nodeIndex = 0; nodeIndex < layerSizes[layerIndex]; nodeIndex ++)
                {
                    for (int weightIndex = 0; weightIndex < layerSizes[layerIndex - 1] + 1; weightIndex ++)
                    {
                        float delta = 0;
                        int weightFlatIndex = getIndex(layerIndex, nodeIndex, weightIndex, maxLayerSize);
                        for (int sampleIndex = 0; sampleIndex < thisBatchNumSamples; sampleIndex ++)
                        {
                            float thisDelta = scratchWeights[sampleIndex * numWeights + weightFlatIndex] - weights[weightFlatIndex];
                            delta += thisDelta;
                        }
                        weights[weightFlatIndex] += delta;
                    }
                }
            }
            if (debug)
            {
                printf("done adding deltas\n");
            }
            for (int i = 0; i < numWeights; i++)
            {
                weightDeltas[i] = 0;
            }
            if (1 || debug)
            {
                printf("Finished epoch %d / %d, batch %d / %d\n",
                    epochIndex, numEpochs,
                    batchNumber, numBatches);
            }
        }
        printf("finished epoch %d\n", epochIndex);
        if (testCases)
        {
            testNetwork(weights, numLayers, layerSizes, testCases);
        }
    }
}

imageTrainingSamples *getImageData(char *filename, int numItems, int startingIndex)
{
    int numInputNodes = 28 * 28;
    int numOutputNodes = 10;

    imageTrainingSamples *samples = (imageTrainingSamples *)malloc(sizeof(imageTrainingSamples));
    samples->inputSamples = (float *)malloc(sizeof(float) * numItems * numInputNodes);
    samples->trueOutput = (float *)malloc(sizeof(float) * numItems * numOutputNodes);

    FILE *fileStream = fopen(filename, "r");

    char *token;
    char line[8192];
    char sep[] = ",";
    int isHeader = 1;
    int itemIndex = 0;
    int numItemsTaken = 0;
    while (numItemsTaken < numItems && fgets(line, 8192, fileStream))
    {
        if (isHeader)
        {
            // skip CSV header
            isHeader = false;
            continue;
        }

        if (itemIndex < startingIndex)
        {
            itemIndex ++;
            continue;
        }

        // first get the label / true value
        token = strtok(line, sep);
        int trueValue = strtol(token, NULL, 10);
        for (int i = 0; i < numOutputNodes; i ++)
        {
            int index = numItemsTaken * numOutputNodes + i;
            if (i == trueValue)
            {
                samples->trueOutput[index] = 1.0f;
            }
            else
            {
                samples->trueOutput[index] = 0.0f;
            }
        }

        // now get the inputs - pixels with value from 0-255
        int tokenIndex = 0;
        while (token != NULL)
        {
            float rawValue = (float)strtol(token, NULL, 10);
            samples->inputSamples[numItemsTaken * numInputNodes + tokenIndex] = rawValue / (float)255;
            token = strtok(NULL, sep);
            tokenIndex ++;
        }
        numItemsTaken ++;
        itemIndex ++;
    }

    samples->numItems = itemIndex;

    return samples;
}

__device__ __host__ void printSampleSketch(float *pixelValues, int sampleIndex)
{
    int width = 28;
    int height = 28;

    char ch = 0;
    float value = 0;

    int sampleOffset = sampleIndex * width * height;

    for (int y = 0; y < height; y ++)
    {
        for (int x = 0; x < width; x ++)
        {
            value = pixelValues[sampleOffset + y * width + x];
            if (value > .75) {
                ch = '#';
            }
            else if (value > .5)
            {
                ch = '=';
            }
            else if (value > .25)
            {
                ch = '-';
            }
            else
            {
                ch = ' ';
            }
            printf("%c", ch);
        }
        printf("\n");
    }
}

int imageSampleTrueValue(float * trueValues, int sampleIndex)
{
    int nodesPerSample = 10;
    int startIndex = nodesPerSample * sampleIndex;

    for (int nodeIndex = 0; nodeIndex < nodesPerSample; nodeIndex ++)
    {
        if (trueValues[startIndex + nodeIndex] == 1)
        {
            return nodeIndex;
        }
    }
    return 0;
}

int imageSampleTestResult(float *trueValues, int sampleIndex, float *result)
{
    int trueValue = imageSampleTrueValue(trueValues, sampleIndex);

    int selectedValue = imageSampleResultToInt(result);

    if (trueValue == selectedValue) {
        return 1;
    }

    return 0;
}

int imageSampleResultToInt(float *result)
{
    int nodesPerSample = 10;
    int selectedValue = 0;
    float highestActivation = 0;
    for (int nodeIndex = 0; nodeIndex < nodesPerSample; nodeIndex ++)
    {
        if (result[nodeIndex] > highestActivation)
        {
            highestActivation = result[nodeIndex];
            selectedValue = nodeIndex;
        }
    }

    return selectedValue;
}

void testNetwork(float *weights, int numLayers, int *layerSizes, imageTrainingSamples *testCases)
{
    int numCorrect = 0;
    for (int testCaseIndex = 0; testCaseIndex < testCases->numItems; testCaseIndex ++)
    {
        int trueValue = imageSampleTrueValue(testCases->trueOutput, testCaseIndex);
        float *result = classify(weights, numLayers, layerSizes, testCases->inputSamples, testCaseIndex);
        int isCorrect = imageSampleTestResult(testCases->trueOutput, testCaseIndex, result);
        // printf("Actual / Result: %d / %d ", trueValue, imageSampleResultToInt(result));
        for (int i = 0; i < layerSizes[numLayers - 1]; i++)
        {
            // printf("%.3f ", result[i]);
        }
        if (isCorrect)
        {
            numCorrect ++;
            // printf("Correct");
        }
        else
        {
            // printf("NOPE");
        }
    }
    printf("Accuracy: %.2f\n", (float)numCorrect / (float) testCases->numItems);
}
